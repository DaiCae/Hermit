
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

double eps = 1e-8;

//检查QR检查
int check_old(double *A, int N){
    for (int x = 0; x < N - 1; x++){
        if (fabs(A[(x + 1) * N + x]) > eps)
            return 1;
    }
    return 0;
}

//检查QR检查
int check(double *A, int N){
    for (int x = 0; x < (N - 1)/2; x++){
        if (fabs(A[(x + 1) * N + x]) > eps)
            return 1;
    }
    return 0;
}

//通过QR(Givens rotation)方法求出全部特征值 A为三对角矩阵 N为矩阵阶数
int QR_OLD(double *A, int N)
{

    double *Q = new double[N * N];
    double *R = new double[N * N];

    for (int k = 0; k < N - 1; k++)
    {
        if (fabs(A[k * N + (k + 1)]) <= eps)
        {
            continue;
        }

        double elem1 = A[k * N + k] * A[k * N + k];
        double elem2 = A[(k + 1) * N + k] * A[(k + 1) * N + k];
        double r = sqrtl(elem1 + elem2);

        //printf("%9lf  %9lf  %9lf\n", A[k*N+k] ,A[(k+1)*N+k],r);
        double Cos = A[k * N + k] / r;
        double Sin = A[(k + 1) * N + k] / r;

        for (int i = 0; i < N; i++)
        {
            for (int j = 0; j < N; j++)
            {
                R[i * N + j] = 0;
                if (i == j)
                    R[i * N + j] = 1;
            }
        }

        R[k * N + k] = Cos;
        R[k * N + (k + 1)] = Sin;
        R[(k + 1) * N + k] = Sin * -1;
        R[(k + 1) * N + (k + 1)] = Cos;

        for (int i = 0; i < N; i++)
            for (int j = 0; j < N; j++)
            {
                Q[i * N + j] = 0.0;
                for (int k = 0; k < N; k++)
                    Q[i * N + j] += R[i * N + k] * A[k * N + j];
            }

        for (int i = 0; i < N; i++)
            for (int j = 0; j < N; j++)
            {
                A[i * N + j] = 0.0;
                for (int k = 0; k < N; k++)
                    A[i * N + j] += Q[i * N + k] * R[k + j * N];
            }
    }

    // printf("\nA 矩阵:\n");
    // for (int i = 0; i < N; i++)
    // {
    //     for (int j = 0; j < N; j++)
    //     {
    //         printf("%9lf  ", A[i * N + j]);
    //     }
    //     printf("\n");
    // }

    delete[] Q;
    delete[] R;

    // return flag;
    return 1;
}

//改进的QR 
int QR_improve(double *A, int N)
{
    double q = (A[N * (N - 2) + N - 2] - A[N * N - 1]) / 2;
    //A[N-2,N-2] A[N-1,N-1]
    double p;
    if (q >= eps)
        p = A[N * N - 1] + q - sqrtl(q * q + A[N * (N - 1) - 1] * A[N * (N - 1) - 1]);
        //A[N-1,N-1] + q - sqrtl(q * q + A[N-2,N-1] * A[N-2.N-1])
    else
        p = A[N * N - 1] + q + sqrtl(q * q + A[N * (N - 1) - 1] * A[N * (N - 1) - 1]);

    // printf("\n%lf,%lf,%lf\n",A[N*(N-2)+N-2], A[N*(N-1)-1], A[N*N-1]);
    // printf("%lf,%lf\n",q,p);

    double *Q = new double[N * N];
    double *R = new double[N * N];
    for (int k = 0; k < (N - 1)/2; k++)
    {
        if (fabs(A[k * N + (k + 1)]) <= eps)
        {
            continue;
        }
        double elem1 = (A[k * N + k] - p) * (A[k * N + k] - p);
        double elem2 = A[(k + 1) * N + k] * A[(k + 1) * N + k];
        double r = sqrtl(elem1 + elem2);

        //printf("%9lf  %9lf  %9lf\n", A[k*N+k] ,A[(k+1)*N+k],r);
        double Cos = (A[k * N + k] - p) / r;
        double Sin = A[(k + 1) * N + k] / r;
        for (int i = 0; i < N; i++)
        {
            for (int j = 0; j < N; j++)
            {
                R[i * N + j] = 0;
                if (i == j)
                    R[i * N + j] = 1;
            }
        }

        R[k * N + k] = Cos;
        R[k * N + (k + 1)] = Sin;
        R[(k + 1) * N + k] = Sin * -1;
        R[(k + 1) * N + (k + 1)] = Cos;
        for (int i = 0; i < N; i++)
            for (int j = 0; j < N; j++)
            {
                Q[i * N + j] = 0.0;
                for (int k = 0; k < N; k++)
                    Q[i * N + j] += R[i * N + k] * A[k * N + j];
            }

        for (int i = 0; i < N; i++)
            for (int j = 0; j < N; j++)
            {
                A[i * N + j] = 0.0;
                for (int k = 0; k < N; k++)
                    A[i * N + j] += Q[i * N + k] * R[k + j * N];
            }
    }

    delete[] Q;
    delete[] R;

    // return flag;
    return 1;
}


int Householder(int n, double A[])
{
	printf("\r开始计算..");
    //使用event计算时间
	hipEvent_t start,stop;
	float elapsedTime = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	//=====================================================================
    double mol, q, value;
    double *alpha = new double[n];
    double *H = new double[n * n];
    double *B = new double[n * n];

    for (int i = n - 1; i > 1; i--)
    {
        q = 0.0;
        //初始化alpha向量为0
        for (int j = 0; j < n; j++)
            alpha[j] = 0.0;

        for (int j = 0; j < i; j++){
            alpha[j] = A[i * n + j];
            q += alpha[j] * alpha[j];
        }
        mol = sqrtl(q);

        if (alpha[i - 1] > 0.0)
            mol = -mol;


        q -= alpha[i - 1] * mol;
        alpha[i - 1] = alpha[i - 1] - mol;

        for (int j = 0; j <= n - 1; j++){
            for (int k = 0; k <= n - 1; k++)
                H[j * n + k] = -alpha[j] * alpha[k] / q;
            H[n * j + j] = H[n * j + j] + 1.0;
        }

        for (int j = 0; j < i+1; j++)
            for (int k = 0; k < i+1; k++)
            {
                value = 0.0;
                for (int u = 0; u < i+1; u++)
                    value = value + H[u + j * n] * A[n * u + k];
                B[k + j * n] = value;
            }

        for (int j = 0; j <  i+1; j++)
            for (int k = 0; k < i+1; k++)
            {
                value = 0.0;
                for (int u = 0; u < i+1; u++)
                    value = value + B[u + j * n] * H[n * u + k];
                A[k + j * n] = value;
            }

    }
    

    delete[] alpha;
    delete[] H;
    delete[] B;

	//=====================================================================
	hipEventRecord(stop,0);
	hipDeviceSynchronize();
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	hipEventDestroy(start);    
	hipEventDestroy(stop);


	printf("\r运行时间:%f(ms)\n",elapsedTime);
    return 1;
}


__global__ void MatMul(double *A, double *B, double *C, int N)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	// printf("A[%2d,%2d] %lf \n",x,y,A[x*N+y]);
	// printf("B[%2d,%2d] %lf \n",x,y,B[x*N+y]);

	double elem1,elem2,value=0;
	for(int k = 0; k < N; k++)
	{

		elem1 = A[x * N + k ];			//取M矩阵的一行
		elem2 = B[k * N + y ];			//取N矩阵的一列
		
		value += elem1 * elem2;	//求和
	}
	C[x * N + y] = value;
}

__global__ void Householder_gpu(double *A, double *B, double * alpha, int N, int k)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	
	//if(x>=N || y>=N) return;
	
	//初始化alpha向量
	if(y==0) 
	{
		alpha[x]=0;
		if(x < k) alpha[x] = A[k * N + x];
	}
	__syncthreads();

	if(x==0 && y==0){
		double mol,sum=0;
		//求出向量mol
		for(int i=0;i<k;i++){
			sum += alpha[i] * alpha[i];
		}
		mol=sqrt(sum);
		//printf("[%10lf]\n",mol);
		if(alpha[k-1] > 0) mol=-mol;
		sum -= alpha[k-1] * mol;
		alpha[k-1]=alpha[k-1] -mol;
		//printf("[%10lf]\n",sum);
		B[0]=sum;
	}	
	
	__syncthreads();
	
	double sum=B[0];
	//计算H矩阵
	B[x * N + y] = -1 * alpha[x] * alpha[y] /sum;
	//对角线元素加一
	if(x==y) B[x * N + y]+=1;

	// if(y==0) printf("%10lf  ",alpha[x]);
	// if(y==0 && x==0) printf("\n");
	// if(y==0 && x==0){
	// 	printf("\nH 矩阵:\n");
    //     for (int i = 0; i < N; i++)
    //     {
    //         for (int j = 0; j < N; j++)
    //         {
    //             printf("%10lf  ", B[i * N + j]);
    //         }
    //         printf("\n");
    //     }
	// }
}

int Householder_cpu(int N, double * Dev_A, double * Dev_B, double * Dev_C )
{	
	printf("开始计算..");
    //使用event计算时间
	hipEvent_t start,stop;
	float elapsedTime = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	//=====================================================================
	double *Dev_alpha;
	hipMalloc((void**)&Dev_alpha, N * sizeof(double));

	for (int k = N - 1; k > 1; k--)
    {
		dim3 block(10,10);
		dim3 grid( N / block.x , N / block.y );
		//dim3 grid(N/block.x);

		//计算H矩阵
		//Householder_gpu<<<grid,block,N * sizeof(double)>>>(Dev_A,Dev_B,N,k);//调用核函数
		Householder_gpu<<<grid,block>>>(Dev_A,Dev_B,Dev_alpha,N,k);//调用核函数

		//A左乘B保存到C
		MatMul<<<grid,block>>>(Dev_A,Dev_B,Dev_C,N);//调用核函数
		MatMul<<<grid,block>>>(Dev_B,Dev_C,Dev_A,N);//调用核函数

		// Householder<<<grid,block>>>(Dev_A,Dev_B,Dev_C,N);//调用核函数
		//break;
	}

	//=====================================================================
	// cudaError_t cudaError = cudaGetLastError();
    // printf("CUDA Errors: %s\n", cudaGetErrorString(cudaError));

	hipEventRecord(stop,0);
	hipDeviceSynchronize();
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	hipEventDestroy(start);    
	hipEventDestroy(stop);
	
	printf("\r运行时间:%f(ms)\n",elapsedTime);
	return 1;
}

__global__ void QR_gpu(double *A, double p, int N ,int k, double eps)
{
	//判断是否为需要的线程块
	// int k = blockIdx.x;
    
    //满足精度要求则终止
    if(fabs(A[(k+1)*N +k]) < eps) return;

    double Cos = A[k * N + k] - p;
    double Sin = A[(k + 1) * N + k];
    double r =sqrt( Cos * Cos + Sin * Sin );
    Cos /= r;
    Sin /= r;


    //取得块矩阵索引
	int x = threadIdx.x;
    int y = threadIdx.y;

    //__shared__ double Sub_A[8];
	__shared__ double Sub_A[4][4];
    __shared__ double Sub_B[4][4];

    //Set Sub_Matrix value
    Sub_A[x][y]=A[(k - 1 + x)*N+(k- 1 + y)];
    Sub_B[x][y]=Sub_A[x][y];
	__syncthreads();

    //Left Mul [Only change two rows]
    if(x==1){
        Sub_B[x][y]=Sub_A[1][y]*Cos + Sub_A[2][y]*Sin;
    }else if(x==2){
        Sub_B[x][y]=Sub_A[1][y]*-1*Sin + Sub_A[2][y]*Cos;
    }
    __syncthreads();

    Sub_A[x][y]=Sub_B[x][y];
    __syncthreads();

    //Right Mul [Only change two columns]
    if(y==1){
        Sub_A[x][y]=Sub_B[x][1]*Cos + Sub_B[x][2]*Sin;
    }else if(y==2){
        Sub_A[x][y]=Sub_B[x][1]*-1*Sin + Sub_B[x][2]*Cos;
    }
    __syncthreads();
    
    // Write result
    A[(k - 1 + x)*N+(k- 1 + y)]=Sub_A[x][y];
	__syncthreads();


}

__global__ void QR_init(double *A, double p, int N , double eps)
{
    if(fabs(A[N]) < eps) return;

    //取得块矩阵索引
	int x = threadIdx.x;
    int y = threadIdx.y;
    
    //shutdown threads
    if(x>2||y>2) return;
    if(x==2 && y==2) return;


    double Cos = A[0] - p;
    double Sin = A[N];
    double r =sqrt( Cos * Cos + Sin * Sin );
    Cos /= r;
    Sin /= r;

    //__shared__ double Sub_A[8];
	__shared__ double Sub_A[3][3];
	__shared__ double Sub_B[3][3];

    //Set Sub_Matrix value
    Sub_A[x][y]=A[x*N+y];
    Sub_B[x][y]=A[x*N+y];
	__syncthreads();

    //Left Mul [Only change two rows]
    if(x==0){
        Sub_B[x][y]=Sub_A[0][y]*Cos + Sub_A[1][y]*Sin;
    }else if(x==1){
        Sub_B[x][y]=Sub_A[0][y]*-1*Sin + Sub_A[1][y]*Cos;
    }
    __syncthreads();

    Sub_A[x][y]=Sub_B[x][y];
    __syncthreads();


    //Right Mul [Only change two columns]
    if(y==0){
        Sub_A[x][y]=Sub_B[x][0]*Cos + Sub_B[x][1]*Sin;
    }else if(y==1){
        Sub_A[x][y]=Sub_B[x][0]*-1*Sin + Sub_B[x][1]*Cos;
    }
    __syncthreads();
    
    // Write result
    A[x*N+y]=Sub_A[x][y];

}

int QR_cpu(int N, double * Dev_A, double * Dev_B, double * Dev_C )
{	
	printf("开始计算..");
    //使用event计算时间
	hipEvent_t start,stop;
	float elapsedTime = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	//=====================================================================
    
	double *elems = new double[3];
	hipMemcpy(elems, Dev_A + ((N-2)*N+(N-2)) , 1 * sizeof(double),hipMemcpyDeviceToHost);
	hipMemcpy(elems + 1, Dev_A + ((N-1)*N+(N-2)) , 2 * sizeof(double),hipMemcpyDeviceToHost);
    //[0]:N-2,N-2
    //[1]:N-1,N-2
    //[2]:N-1,N-1;
    
    printf("CPU: %lf ,%lf ,%lf \n",elems[0],elems[1],elems[2]);
    
    double q = (elems[0] - elems[2]) / 2;
    double p = 0;
    if(q>=0)
        p=elems[2] + q - sqrtl( q * q + elems[1] * elems[1]);
    else
        p=elems[2] + q + sqrtl( q * q + elems[1] * elems[1]);
    printf("q:%lf, p:%lf\n",q,p);
    
    
    dim3 grid(1);
    dim3 block(4,4);

    //QR_init<<<grid,block>>>(Dev_A,p,N,eps);

    for(int i=0;i<500;i++){
        QR_init<<<grid,block>>>(Dev_A,p,N,eps);
        for (int k = 1; k < (N-1)*0.5; k++)
        {
            QR_gpu<<<grid,block>>>(Dev_A,p,N,k,eps);
        }
    }


	//=====================================================================
	hipError_t hipError_t = hipGetLastError();
    printf("CUDA Errors: %s\n", hipGetErrorString(hipError_t));

	hipEventRecord(stop,0);
	hipDeviceSynchronize();
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime,start,stop);
	hipEventDestroy(start);    
	hipEventDestroy(stop);
	
	printf("\r运行时间:%f(ms)\n",elapsedTime);
	return 1;
}

int main()
{
	printf("分配CPU内存空间..\n");
    //double *A,*B,*C;


	// int N = 4;
    // double A[16] = {
	// 	4.0, 1.0, -2.0, 2.0, 
	// 	1.0, 2.0, 0.0, 1.0,
	// 	-2.0, 0.0, 3.0, -2.0,
	// 	2.0, 1.0, -2.0, -1.0
	// };

	int N=10;
    double A[100] = {1.0, 2.0, 3.0, 4.0, 5.0, 0.0, -1.0, -1.0, -1.0, -1.0,
                     2.0, 2.0, 3.0, 4.0, 6.0, 1.0, 0.0, 1.0, 0.0, 0.0,
                     3.0, 3.0, 3.0, 1.0, 5.0, 1.0, -1.0, 0.0, 0.0, 0.0,
                     4.0, 4.0, 1.0, 3.0, 1.0, 1.0, 0.0, 0.0, 0.0, -1.0,
                     5.0, 6.0, 5.0, 1.0, 3.0, 1.0, 0.0, 0.0, 1.0, 0.0,
                     0.0, 1.0, 1.0, 1.0, 1.0, 1.0, 2.0, 3.0, 4.0, 5.0,
                     -1.0, 0.0, -1.0, 0.0, 0.0, 2.0, 2.0, 3.0, 4.0, 6.0,
                     -1.0, 1.0, 0.0, 0.0, 0.0, 3.0, 3.0, 3.0, 1.0, 5.0,
                     -1.0, 0.0, 0.0, 0.0, 1.0, 4.0, 4.0, 1.0, 3.0, 1.0,
                     -1.0, 0.0, 0.0, -1.0, 0.0, 5.0, 6.0, 5.0, 1.0, 3.0};



	
	double *B = new double[N * N];
    double *C = new double[N * N];
	
	printf("分配GPU内存空间..\n");
	//定义GPU内存指针
	double *Dev_A,*Dev_B,*Dev_C;
	//设备端内存分配
	hipMalloc((void**)&Dev_A, N * N * sizeof(double));
	hipMalloc((void**)&Dev_B, N * N * sizeof(double));
	hipMalloc((void**)&Dev_C, N * N * sizeof(double));


	printf("内存拷贝..\n");
	//数据拷贝，主机到设备
	hipMemcpy(Dev_A, A, N * N * sizeof(double),hipMemcpyHostToDevice);
    
	//QR计数
	int num = 0;

	printf("\n========================GPU=======================\n");

	double *A0 = new double[N * N];
	double *B0 = new double[N * N];	
    
    Householder_cpu(N,Dev_A,Dev_B,Dev_C);
	hipMemcpy(A0, Dev_A, N * N * sizeof(double),hipMemcpyDeviceToHost);
	
	QR_cpu(N,Dev_A,Dev_B,Dev_C);
	hipMemcpy(B0, Dev_A, N * N * sizeof(double),hipMemcpyDeviceToHost);

	printf("\nHouseholder 矩阵:\n");
	for (int i = 0; i < N; i++){
		for (int j = 0; j < N; j++)
			printf("%11lf,", A0[i * N + j]);
		printf("\n");
	}

    // while (check(A0, N)){
    //     QR_improve(A0, N);
    //     printf("\rNo.%d", ++num);
    // }
    // printf("\rTotal loop(QR):%d \n", num);
	printf("\nQR 矩阵:\n");
	for (int i = 0; i < N; i++){
		for (int j = 0; j < N; j++)
            printf("%11.7lf,", B0[i * N + j]);
		printf("\n");
	}

	// printf("\n========================CPU=======================\n");
	// Householder(N,A);
	// printf("\nCPU 矩阵:\n");
	// for (int i = 0; i < N; i++){
	// 	for (int j = 0; j < N; j++)
	// 		printf("%11lf  ", A[i * N + j]);
	// 	printf("\n");
	// }

	// num=0;
    // while (check_old(A, N)){
    //     QR_OLD(A, N);
    //     printf("\rNo.%d", ++num);
    // }
    // printf("\rTotal loop(QR):%d \n", num);

	// for (int i = 0; i < N; i++){
	// 	for (int j = 0; j < N; j++)
	// 		printf("%11lf  ", A[i * N + j]);
	// 	printf("\n");
	// }




	//释放GPU内存
	hipFree(Dev_A);
	hipFree(Dev_B);
	hipFree(Dev_C);


	
	//释放CPU内存
    // free(A);
    // free(B);
	free(C);

	return 0;
}


